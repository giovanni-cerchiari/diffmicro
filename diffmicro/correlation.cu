#include "hip/hip_runtime.h"
/*
Copyright: Giovanni Cerchiari
e-mail: giovanni.cerchiari@gmail.com
date: 8/2011
*/

/*
Copyright: Mojtaba Norouzisadeh, Giovanni Cerchiari
e-mail: giovanni.cerchiari@gmail.com
e-mail: norouzi.mojtaba.sade@gmail.com

update: 05/2020 - 09/2020
*/


/*
This file is part of diffmicro.

    Diffmicro is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    Diffmicro is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with Diffmicro.  If not, see <https://www.gnu.org/licenses/>.
*/


/*!
This functions are written for diffmicro.exe application.
*/

#include "correlation.h"
#include "figure_opencv.h"
#include ""

window_display_opencv *fft_window(NULL);
window_display_opencv *image_window(NULL);
double *image_l(NULL);
double *module_fft(NULL);
double *phase_fft(NULL);
unsigned __int16 *tmp_display_i16(NULL);
CUFFT_COMPLEX *tmp_display_cpx(NULL);

sizes s_load_image;
sizes s_radial_lut;
sizes s_fft;
sizes s_fft_images;
sizes s_power_spectra;
sizes s_time_series;
sizes s_fft_time;


unsigned short *dev_im_gpu(NULL);
unsigned int *dev_radial_lut_gpu(NULL);
CUFFT_COMPLEX *dev_fft_gpu(NULL);
CUFFT_COMPLEX* dev_fft_time_gpu(NULL);
CUFFT_REAL* dev_corr_gpu(NULL);
CUFFT_COMPLEX *dev_images_gpu(NULL);
CUFFT_COMPLEX *dev_image_sot_gpu(NULL);
STORE_REAL *dev_power_spectra_gpu(NULL);

//Mohammed
//CUFFT_COMPLEX* dev_fft_gpu_(NULL);
STORE_REAL* dev_ALLfft_diff(NULL);
STORE_REAL* dev_ALLpower_spectra(NULL);

CUFFT_REAL* dev_corr_gpu1(NULL);
CUFFT_COMPLEX* dev_fft_time_gpu1(NULL);
CUFFT_COMPLEX* dev_images_gpu1(NULL);



//! this is the inverse norm of the FFT to have the operation normalized
CUFFT_REAL one_over_fft_norm;
//! CUDA variable necessary to calculate a FFT
hipfftHandle plan;
hipfftHandle tplan;


/*!
from image in unsigned short format to a CUFFT_COMPLEX memory area where the FFT will be calculated
*/
__global__ void short_to_real_with_gain(INDEX dim, unsigned short in[], CUFFT_REAL gain, CUFFT_COMPLEX out[])
{
	INDEX i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < dim)
	{
		out[i].x = gain * (CUFFT_REAL)(in[i]);
		out[i].y = 0.;

		//std::cout << out[i].x << std::endl;
	}
}

/*!
Gain of a complex array by a real coefficient
*/
__global__ void gain_complex(CUFFT_REAL gain, INDEX dim, CUFFT_COMPLEX in[], CUFFT_COMPLEX out[])
{
	INDEX i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < dim)
	{
		out[i].x = gain * in[i].x;
		out[i].y = gain * in[i].y;
	}
}


/*!This kernel copy with real gain and lut*/
__global__ void gain_complex_lut(CUFFT_REAL gain, INDEX dim, unsigned int *lut, CUFFT_COMPLEX in[], CUFFT_COMPLEX out[])
{
	INDEX i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < dim)
	{
		out[i].x = gain * in[lut[i]].x;
		out[i].y = gain * in[lut[i]].y;
	}
}

__global__ void gain_complex_lut_timeSeries(int ii,INDEX nimages, CUFFT_REAL gain, INDEX dim, unsigned int* lut, CUFFT_COMPLEX in[], CUFFT_COMPLEX out[])
{
	INDEX i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < dim)
	{
		out[i* nimages+ii].x = gain * in[lut[i]].x;
		out[i* nimages+ii].y = gain * in[lut[i]].y;
	}
}


/*!
This kernel executes the difference between the upper half of two Fourier transforms, calulate the power spectrum
and stores the result refreshing the values o a power spectrum out[]. coef1 and coef2 are the refreshing coefficient
used to make a linear combination.
*/
__global__ void diff_power_spectrum_to_avg_gpu
  (INDEX dim, CUFFT_COMPLEX min[], CUFFT_COMPLEX sot[], CUFFT_REAL coef1, CUFFT_REAL coef2, STORE_REAL out[])
{
	INDEX i = blockDim.x * blockIdx.x + threadIdx.x;
	CUFFT_REAL difx, dify;

	if (i < dim)
	{
		difx = min[i].x - sot[i].x;
		dify = min[i].y - sot[i].y;
		out[i] = coef1 * out[i] + coef2 * (difx * difx + dify * dify);
	}
}

__global__ void averagesabs2_array_gpu(INDEX dim, INDEX dim_t, CUFFT_COMPLEX* _in, CUFFT_REAL* out)
{
	INDEX j = blockDim.x * blockIdx.x + threadIdx.x;
	// Does the time series exists?
	if (j < dim_t)
	{
		CUFFT_COMPLEX* in;
		// selection of the time series
		in = &(_in[j * dim]);

		FFTW_REAL avg = 0.0;
		FFTW_REAL coef1, coef2, abs2_fromstart, abs2_fromend;
		INDEX i, ii;
		for (i = 0; i < dim; ++i)
		{
			// next absolute value from the beginning of the array
			abs2_fromstart = in[i ].x * in[i ].x + in[i ].y * in[i ].y;

			// next absolute value from the end of the array
			ii = dim - 1 - i;
			abs2_fromend = in[ii ].x * in[ii ].x + in[ii ].y * in[ii ].y;

			// in-place average
			coef2 = (FFTW_REAL)(1.0) / (FFTW_REAL)(i + 1);
			coef1 = (FFTW_REAL)(i)*coef2;
			avg = coef1 * avg + coef2 * (abs2_fromstart + abs2_fromend);

			// save the result in the output array.
			// This operation must be done inside the for loop.
			// ATTENTION! note the index
			out[ii + j * dim] = avg;
		}
	}
}



__global__ void gaincomplex_gpu(INDEX dim, CUFFT_COMPLEX in[], FFTW_REAL gain, CUFFT_COMPLEX out[])
{
	INDEX i = blockDim.x * blockIdx.x + threadIdx.x;

	if ( i < dim)
	{
		out[i].x = gain * in[i].x;
		out[i].y = gain * in[i].y;
	}
}

__global__ void gaincomplex_gpu2(INDEX nimages,INDEX fft_size,INDEX N2, CUFFT_COMPLEX* in, FFTW_REAL gain, CUFFT_COMPLEX* out)
{
	INDEX j = blockDim.y * blockIdx.y + threadIdx.y;

	INDEX i = blockDim.x * blockIdx.x + threadIdx.x;

	if ((i < nimages)&&(j< fft_size))
	{
		out[i+j*N2].x = gain * in[i+j* nimages].x;
		out[i+j*N2].y = gain * in[i+j* nimages].y;
	}
}

__global__ void complexabs2_gpu(INDEX dim, CUFFT_COMPLEX* in, CUFFT_COMPLEX* out)
{
	INDEX i = blockDim.x * blockIdx.x + threadIdx.x;

	if  ( i < dim)
	{
		out[i].x = in[i].x * in[i].x + in[i].y * in[i].y;
		out[i].y = 0.0;
	}
}

__global__ void updatewithdivrebyramp_gpu(INDEX dim, INDEX ramp_start, CUFFT_COMPLEX* in, FFTW_REAL* update)
{
	INDEX i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < dim)
		update[i] -= (2. / (FFTW_REAL)(ramp_start - i)) * in[i].x;
}

__global__ void updatewithdivrebyramp_gpu2(INDEX nimages, INDEX fft_size, INDEX N2, CUFFT_COMPLEX* in, FFTW_REAL* update, STORE_REAL* dev_power_spectra_gpu)
{
	INDEX j = blockDim.y * blockIdx.y + threadIdx.y;

	INDEX i = blockDim.x * blockIdx.x + threadIdx.x;

	if ((i < nimages) && (j < fft_size)) {
		update[i + j * nimages] -= (2. / (FFTW_REAL)(nimages - i)) * in[i + j * N2].x;
		dev_power_spectra_gpu[i * nimages + j] = update[i + j * nimages];
	}

}


__global__ void copyfrom_gpu(INDEX dim, CUFFT_COMPLEX* tseries, FFTW_REAL* corr_memory)
{

	INDEX i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < dim)
	{
		
		tseries[i ].x = corr_memory[i ];
		tseries[i ].y = 0.0;
	}
}


__global__ void cpx_col2row_gain_gpu(INDEX dimcopy, INDEX dimx_in, INDEX i_col_in, CUFFT_COMPLEX in[], CUFFT_REAL gain, INDEX dimx_out, INDEX i_row_out, CUFFT_COMPLEX out[])
{

	INDEX i = blockDim.x * blockIdx.x + threadIdx.x;


	INDEX i_in, i_out;

	if (i < dimcopy)
	{
		i_in = i * dimx_in + i_col_in;
		i_out = i_row_out * dimx_out + i;

		out[i_out].x = gain * in[i_in].x;
		out[i_out].y= gain * in[i_in].y;
	}
}


__global__ void cpx_row2col_gain_gpu(INDEX dim, INDEX dimx_in, INDEX i_row_in, CUFFT_COMPLEX in[], FFTW_REAL gain, INDEX dimx_out, INDEX i_col_out, CUFFT_COMPLEX out[])
{
	INDEX i = blockDim.x * blockIdx.x + threadIdx.x;
	INDEX i_in, i_out;
	if (i < dim)
	{
		i_in = i_row_in * dimx_in + i;
		i_out = i * dimx_out + i_col_out;

		out[i_out].x = gain * in[i_in].x;
		out[i_out].y = gain * in[i_in].y;
	}
}


__global__ void complextorealwithgain_gpu(INDEX dim, CUFFT_COMPLEX vets[], CUFFT_REAL gain, CUFFT_REAL vetc[])
{
	INDEX i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < dim)
	{
		vetc[i] = gain * vets[i].x;
	}
}

__global__ void fft_diff(int z, INDEX fft_size, INDEX nb_fft, CUFFT_COMPLEX ALLfft[], STORE_REAL ALLfft_diff[])
{
	int ii = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	CUFFT_REAL difx, dify;
	//for (int ii = 0; ii < nb_fft - 1; ii++) {
	   // for (int j = 0; j < fft_size; j++) {
	if ((ii < nb_fft - 1 - z) && (j < fft_size)) {

		difx= ALLfft[j + ii * fft_size].x - ALLfft[j + fft_size + ii * fft_size + z * fft_size].x;
		dify= ALLfft[j + ii * fft_size].y - ALLfft[j + fft_size + ii * fft_size + z * fft_size].y;

		ALLfft_diff[j + ii * fft_size] = difx*difx + dify*dify;
	}
	//std::cout << ALLfft_diff[j + ii * fft_size] << "   ";
// }
 //std::cout << std::endl;
//}
}

__global__ void structure_function(int z, INDEX fft_size, INDEX nb_fft, STORE_REAL ALLfft_diff[], STORE_REAL power_spectra[])
{
	//int z = blockIdx.y * blockDim.y + threadIdx.y;

	int k = blockIdx.x * blockDim.x + threadIdx.x;

	/*tot = 0;
	for (int len = 1; len <= z; len++) {
		tot += fft_size * (nb_fft - len);
	}*/

	double norm = 1. / (nb_fft - (z + 1));

	if ((k < fft_size) && (z < nb_fft - 1)) {
		for (int kk = 0; kk < nb_fft - 1 - z; kk++) {
			power_spectra[k + z * fft_size] += norm*ALLfft_diff[k + kk * fft_size];
		}
	}

}

void time_series_analysis_gpu() {

	cuda_exec mycuda_dim_t, mycuda_dim, mycuda_dim_dim_t;

	lldiv_t group;
	INDEX i, n_group, group_rem;

	time_time_correlation.start();
	
	calc_cuda_exec(s_time_series.dim, deviceProp.maxThreadsPerBlock, &mycuda_dim);
	calc_cuda_exec(useri.nthread_gpu, deviceProp.maxThreadsPerBlock, &mycuda_dim_t);
	calc_cuda_exec(s_time_series.dim * useri.nthread_gpu, deviceProp.maxThreadsPerBlock, &mycuda_dim_dim_t);


	group = std::div((long long)(s_time_series.numerosity), (long long)(useri.nthread_gpu));
	n_group = (INDEX)(group.quot);
	group_rem = (INDEX)(group.rem);

	//-----------------------------------------
	// NO GROUP SPLIT, IT IS ASSUMED THAT MEMORY WILL BE ENOUGH... TO BE CHANGED

	for (i = 0; i < n_group; ++i)
	{
		//std::cout << i << std::endl;
		timeseriesanalysis_gpu(s_time_series.dim, useri.nthread_gpu , &dev_images_gpu[i*useri.nthread_gpu* s_time_series.dim], s_fft_time.dim, dev_fft_time_gpu, &tplan, dev_corr_gpu,
			mycuda_dim_t, mycuda_dim, mycuda_dim_dim_t);

		
		
	}
	if (0 != group_rem)
	{
		calc_cuda_exec(group_rem, deviceProp.maxThreadsPerBlock, &mycuda_dim_t);
		calc_cuda_exec(s_time_series.dim * group_rem, deviceProp.maxThreadsPerBlock, &mycuda_dim_dim_t);
		timeseriesanalysis_gpu(s_time_series.dim, group_rem, &dev_images_gpu[i * useri.nthread_gpu * s_time_series.dim], s_fft_time.dim, dev_fft_time_gpu, &tplan, dev_corr_gpu,
			mycuda_dim_t, mycuda_dim, mycuda_dim_dim_t);

	}

	time_time_correlation.stop();

	time_from_device_to_host.start();
	hipMemcpy(dev_images_cpu, dev_images_gpu, s_time_series.memory_tot, hipMemcpyDeviceToHost);
	time_from_device_to_host.stop();

	/*timeseriesanalysis_gpu(s_time_series.dim, s_time_series.numerosity, dev_images_gpu, s_fft_time.dim, dev_fft_time_gpu, &tplan, dev_corr_gpu,
		mycuda_dim_t, mycuda_dim, mycuda_dim_dim_t);*/

	//timeseriesanalysis_gpu(dim, dim_tseries, yinputg2, dimp, yg2, &plan, out_f2_g, mycuda_dim_t, mycuda_dim, mycuda_dim_dim_t);
	
}

void timeseries_to_lutpw_gpu(INDEX dimcopy, CUFFT_REAL gain, INDEX t, INDEX starting_freq, STORE_REAL* ram_power_spectra) {

	cuda_exec mycuda_dim_ii;
	calc_cuda_exec(dimcopy, deviceProp.maxThreadsPerBlock, &mycuda_dim_ii);

	timeseries_to_lutfft_gpu(dimcopy, gain, t, starting_freq);
	//complex_to_real_with_gain_cpu(useri.nthread, dimcopy, dev_image_sot_cpu,
	//		(FFTW_REAL)(1.0), dev_power_spectra_cpu);
	complextorealwithgain_gpu<<<mycuda_dim_ii.nbk, mycuda_dim_ii.nth >>>
		( dimcopy, dev_image_sot_gpu,(FFTW_REAL)(1.0), dev_power_spectra_gpu);

	hipMemcpy(ram_power_spectra, dev_power_spectra_gpu, sizeof(CUFFT_REAL)* dimcopy, hipMemcpyDeviceToHost);



}
void timeseries_to_lutfft_gpu(INDEX dimcopy, CUFFT_REAL gain, INDEX t, INDEX starting_freq) {

	cuda_exec mycuda_dim_ii;
	calc_cuda_exec(dimcopy, deviceProp.maxThreadsPerBlock, &mycuda_dim_ii);

	cpx_col2row_gain_gpu<<<mycuda_dim_ii.nbk, mycuda_dim_ii.nth >>>(dimcopy, s_time_series.dim, t, dev_images_gpu, gain,
		(INDEX)(1), starting_freq, dev_image_sot_gpu);


}


void lutfft_to_timeseries_gpu(INDEX dimcopy, CUFFT_REAL gain, INDEX t, INDEX starting_freq) {

	cuda_exec mycuda_dim_i;
	calc_cuda_exec(dimcopy, deviceProp.maxThreadsPerBlock, &mycuda_dim_i);

	cpx_row2col_gain_gpu<<<mycuda_dim_i.nbk, mycuda_dim_i.nth >>>(dimcopy, (INDEX)(1), starting_freq, dev_image_sot_gpu, gain,
		s_time_series.dim, t, dev_images_gpu);

}

/*
void number_bk_th(INDEX dim, unsigned int n_max_th, unsigned int &nbk, unsigned int &nth)
{

	if(dim > n_max_th)
		{
			nth = n_max_th;
			nbk = dim / n_max_th;
		}
	else
		{
			nth = dim;
			nbk = 1;
		}
}*/

void calc_sizes(INDEX dimy, INDEX dimx, INDEX n_tot, INDEX size_of_element, sizes &s)
{
	s.dimx = dimx;
	s.dimy = dimy;
	s.numerosity = n_tot;
	
	s.dim = dimx * dimy;
	s.memory_one = s.dim * size_of_element;
	s.memory_tot = n_tot * s.memory_one;
	calc_cuda_exec(s.dim, deviceProp.maxThreadsPerBlock, &s.cexe);
}


int gpu_allocation(int flg_mode, INDEX &nimages, INDEX &dimy, INDEX &dimx, INDEX &dim_power_spectrum, unsigned int *ram_radial_lut)
{
	int alloc_status_li, alloc_status_fft, alloc_status_pw, alloc_status_im, alloc_status_plan , alloc_status_plan_time, alloc_status_rlut, alloc_status_imsot;
	int alloc_status_fftime, alloc_status_corr_g;
	INDEX i, free_video_memory, image_p_spectrum_memory;
	INDEX capacity;

	INDEX dimtimeseries_exponent, dimtimeseries_zeropadding;
	double capacity_d;

	calc_sizes(dimy, dimx, 1, sizeof(unsigned short), s_load_image);
	calc_sizes(1, dim_power_spectrum, 1, sizeof(unsigned int), s_radial_lut);
	calc_sizes(dimy, dimx, 1, sizeof(CUFFT_COMPLEX), s_fft);
	calc_sizes(1, dim_power_spectrum, 1, sizeof(CUFFT_COMPLEX), s_fft_images);
	calc_sizes(1, dim_power_spectrum, 1, sizeof(STORE_REAL), s_power_spectra);
	calc_sizes(1, nimages, 1, sizeof(CUFFT_COMPLEX), s_time_series);
	// Calculating the closes power of 2 that allows for zero-padding
	dimtimeseries_exponent = (INDEX)(std::ceil(std::log((FFTW_REAL)(nimages)) / std::log(2.0))) + 1;
	dimtimeseries_zeropadding = 1;
	for (i = 0; i < dimtimeseries_exponent; ++i) dimtimeseries_zeropadding *= 2;
	calc_sizes(1, dimtimeseries_zeropadding, 1, sizeof(CUFFT_COMPLEX), s_fft_time);

	// extimating video card capability of storing the same number of power spectra and
	// FFT of different images
	free_video_memory = (INDEX)(deviceProp.totalGlobalMem) - (s_load_image.memory_tot + s_fft.memory_tot + s_radial_lut.memory_tot + s_fft_images.memory_one);
	

	switch (useri.execution_mode)
	{
	case DIFFMICRO_MODE_FIFO:
		image_p_spectrum_memory = s_power_spectra.memory_one + s_fft_images.memory_one;

		capacity = free_video_memory / image_p_spectrum_memory;
		if (capacity > nimages) capacity = nimages;
		break;
	case DIFFMICRO_MODE_TIMECORRELATION:
		//memory.tot
		free_video_memory -= s_fft_time.memory_one * useri.nthread_gpu;
		image_p_spectrum_memory = s_time_series.memory_one;
		capacity = free_video_memory / image_p_spectrum_memory;
		if (capacity > s_fft_images.dim) capacity = s_fft_images.dim;
		break;
	default:
		std::cerr << "invalid diffmicro mode" << std::endl;
		return 1;
		break;
	}



	if(capacity == 0)
		{
			std::cerr <<"not enough video card memory for this task"<<std::endl;
			return false;
		}

		
	one_over_fft_norm = (CUFFT_REAL)(1./(sqrt((CUFFT_REAL)(dimx * dimy))));
	gpu_free_pointers();

	//----------------------------------------------------------------------------
	// ALLOCATION
	
	// trial and error allocation

	//----------------------------------------------------------
	// CUFFT initialization
#if (CUFFT_TYPE == CUFFT_TYPE_FLOAT)
	alloc_status_plan = hipfftPlan2d(&plan, dimy, dimx, HIPFFT_C2C);
	//hipfftExecC2C(plan, dev_fft, dev_fft, HIPFFT_FORWARD);
#elif (CUFFT_TYPE == CUFFT_TYPE_DOUBLE)
	alloc_status_plan = hipfftPlan2d(&plan, dimy, dimx, HIPFFT_Z2Z);
	//hipfftExecZ2Z(plan, dev_fft, dev_fft, HIPFFT_FORWARD);
#else
#error Unknown CUDA type selected
#endif

	if (hipSuccess != alloc_status_plan)
	{
		std::cerr << "cuda error in inizializing plan for FFT" << std::endl;
		return 1;
	}

	switch (useri.execution_mode)
	{
	case DIFFMICRO_MODE_FIFO:


		alloc_status_im = hipMalloc(&dev_images_gpu, s_fft_images.memory_one * capacity );
		alloc_status_pw = hipMalloc(&dev_power_spectra_gpu, s_power_spectra.memory_one * capacity );
		alloc_status_fft = hipMalloc(&dev_fft_gpu, s_fft.memory_tot );
		alloc_status_li = hipMalloc(&dev_im_gpu, s_load_image.memory_tot );
		alloc_status_rlut = hipMalloc(&dev_radial_lut_gpu, s_radial_lut.memory_tot );
		alloc_status_imsot = hipMalloc(&dev_image_sot_gpu, s_fft_images.memory_one);
		capacity_d = (double)(capacity);
		while((hipSuccess != alloc_status_pw) || (hipSuccess != alloc_status_im) ||
					 (hipSuccess != alloc_status_li) || (hipSuccess != alloc_status_fft) ||
								(HIPFFT_SUCCESS != alloc_status_plan) || (hipSuccess != alloc_status_rlut) || (hipSuccess != alloc_status_imsot))
			{
				//printf("capacity %u\r\n", capacity);
				capacity_d *= 0.95;
				capacity_d = std::floor( capacity_d * 0.95);
				capacity = (INDEX)(capacity_d);
				if(HIPFFT_SUCCESS == alloc_status_plan) hipfftDestroy(plan);
				if(hipSuccess == alloc_status_pw) hipFree(dev_power_spectra_gpu);
				if(hipSuccess == alloc_status_im) hipFree(dev_images_gpu);
				if(hipSuccess == alloc_status_li) hipFree(dev_im_gpu);
				if(hipSuccess == alloc_status_fft) hipFree(dev_fft_gpu);
				if(hipSuccess == alloc_status_rlut) hipFree(dev_radial_lut_gpu);
				if(hipSuccess == alloc_status_imsot) hipFree(dev_image_sot_gpu);
				//----------------------------------------------------------
				// CUFFT initialization
				#if (CUFFT_TYPE == CUFFT_TYPE_FLOAT)
					alloc_status_plan = hipfftPlan2d(&plan, dimy, dimx, HIPFFT_C2C);
					//hipfftExecC2C(plan, dev_fft, dev_fft, HIPFFT_FORWARD);
				#elif (CUFFT_TYPE == CUFFT_TYPE_DOUBLE)
					alloc_status_plan = hipfftPlan2d(&plan, dimy, dimx, HIPFFT_Z2Z);
					//hipfftExecZ2Z(plan, dev_fft, dev_fft, HIPFFT_FORWARD);
				#else
					#error Unknown CUDA type selected
				#endif

				alloc_status_im = hipMalloc(&dev_images_gpu, s_fft_images.memory_one * capacity );
				alloc_status_pw = hipMalloc(&dev_power_spectra_gpu, s_power_spectra.memory_one * capacity );
				alloc_status_fft = hipMalloc(&dev_fft_gpu, s_fft.memory_tot );
				alloc_status_li = hipMalloc(&dev_im_gpu, s_load_image.memory_tot );
				alloc_status_rlut = hipMalloc(&dev_radial_lut_gpu, s_radial_lut.memory_tot);
				alloc_status_imsot = hipMalloc(&dev_image_sot_gpu, s_fft_images.memory_one);

			}
		


		//----------------------------------------------------------
		//----------------------------------------------------------
		//----------------------------------------------------------
		// this operation that seems unuseful in fact reset the plan in a such a way that, if ever some allocation errors
		// as occured, the program will work properly
		// this fact is purely experimental and I cannot explain why it happens!!!
				// CUFFT initialization
				#if (CUFFT_TYPE == CUFFT_TYPE_FLOAT)
					hipfftExecC2C(plan, dev_fft, dev_fft, HIPFFT_FORWARD);
				#elif (CUFFT_TYPE == CUFFT_TYPE_DOUBLE)
					hipfftExecZ2Z(plan, dev_fft_gpu, dev_fft_gpu, HIPFFT_FORWARD);
				#else
					#error Unknown CUDA type selected
				#endif
		hipDeviceSynchronize();

		//----------------------------------------------------------
		//----------------------------------------------------------
		//----------------------------------------------------------


		calc_sizes(1, dim_power_spectrum, capacity, sizeof(CUFFT_COMPLEX), s_fft_images);
		tot_memory_fft_images = s_fft_images.memory_tot;

		calc_sizes(1, dim_power_spectrum, capacity, sizeof(STORE_REAL), s_power_spectra);
		tot_calculation_memory = s_power_spectra.memory_tot;
		hipMemcpy(dev_radial_lut_gpu, ram_radial_lut, s_radial_lut.memory_one, hipMemcpyHostToDevice);
		break;

	case DIFFMICRO_MODE_TIMECORRELATION:
	{
		

		int n[1] = { s_fft_time.dim };

#if (CUFFT_TYPE == CUFFT_TYPE_FLOAT)
		alloc_status_plan_time = hipfftPlanMany(&tplan, 1, n,
			NULL, 1, s_fft_time.dim,  //advanced data layout, NULL shuts it off
			NULL, 1, s_fft_time.dim,  //advanced data layout, NULL shuts it off
			HIPFFT_C2C, useri.nthread_gpu);
		//hipfftExecC2C(tplan, dev_fft, dev_fft, HIPFFT_FORWARD);
#elif (CUFFT_TYPE == CUFFT_TYPE_DOUBLE)
		alloc_status_plan_time = hipfftPlanMany(&tplan, 1, n,
			NULL, 1, s_fft_time.dim,  //advanced data layout, NULL shuts it off
			NULL, 1, s_fft_time.dim,  //advanced data layout, NULL shuts it off
			HIPFFT_Z2Z, useri.nthread_gpu);
		//hipfftExecZ2Z(tplan, dev_fft, dev_fft, HIPFFT_FORWARD);
#else
#error Unknown CUDA type selected
#endif
		hipDeviceSynchronize();

		if (hipSuccess != alloc_status_plan_time)
		{
			std::cerr << "cuda error in inizializing plan for FFT" << std::endl;
			return 1;
		}

		
	

		alloc_status_im = hipMalloc(&dev_images_gpu, s_time_series.memory_one * capacity);
		alloc_status_fftime = hipMalloc(&dev_fft_time_gpu, s_fft_time.memory_one * useri.nthread_gpu);
		alloc_status_corr_g = hipMalloc(&dev_corr_gpu, s_time_series.memory_one * useri.nthread_gpu);
		alloc_status_pw = hipMalloc(&dev_power_spectra_gpu, s_power_spectra.memory_one );
		alloc_status_fft = hipMalloc(&dev_fft_gpu, s_fft.memory_tot);
		alloc_status_li = hipMalloc(&dev_im_gpu, s_load_image.memory_tot);
		alloc_status_rlut = hipMalloc(&dev_radial_lut_gpu, s_radial_lut.memory_tot);
		alloc_status_imsot = hipMalloc(&dev_image_sot_gpu, s_fft_images.memory_one);


		
		

		capacity_d = (double)(capacity);

		while ((hipSuccess != alloc_status_pw) || (hipSuccess != alloc_status_im) ||
			(hipSuccess != alloc_status_li) || (hipSuccess != alloc_status_fft) ||
			(HIPFFT_SUCCESS != alloc_status_plan) || (hipSuccess != alloc_status_rlut) || (hipSuccess != alloc_status_imsot) ||
			 (hipSuccess != alloc_status_fftime) || (hipSuccess != alloc_status_corr_g) )
		{
			//printf("capacity %u\r\n", capacity);
			capacity_d *= 0.95;
			capacity_d = std::floor(capacity_d * 0.95);
			capacity = (INDEX)(capacity_d);
			if (HIPFFT_SUCCESS == alloc_status_plan) hipfftDestroy(tplan);
			if (hipSuccess == alloc_status_pw) hipFree(dev_power_spectra_gpu);
			if (hipSuccess == alloc_status_im) hipFree(dev_images_gpu);
			if (hipSuccess == alloc_status_li) hipFree(dev_im_gpu);
			if (hipSuccess == alloc_status_fft) hipFree(dev_fft_gpu);
			if (hipSuccess == alloc_status_rlut) hipFree(dev_radial_lut_gpu);
			if (hipSuccess == alloc_status_imsot) hipFree(dev_image_sot_gpu);
			if (hipSuccess == alloc_status_fftime) hipFree(dev_fft_time_gpu);
			if (hipSuccess == alloc_status_corr_g) hipFree(dev_corr_gpu);

			//----------------------------------------------------------
			// CUFFT initialization
#if (CUFFT_TYPE == CUFFT_TYPE_FLOAT)
			alloc_status_plan_time = hipfftPlanMany(&tplan, 1, n,
				NULL, 1, s_fft_time.dim,  //advanced data layout, NULL shuts it off
				NULL, 1, s_fft_time.dim,  //advanced data layout, NULL shuts it off
				HIPFFT_C2C, useri.nthread_gpu);
			//hipfftExecC2C(tplan, dev_fft, dev_fft, HIPFFT_FORWARD);
#elif (CUFFT_TYPE == CUFFT_TYPE_DOUBLE)
			alloc_status_plan_time = hipfftPlanMany(&tplan, 1, n,
				NULL, 1, s_fft_time.dim,  //advanced data layout, NULL shuts it off
				NULL, 1, s_fft_time.dim,  //advanced data layout, NULL shuts it off
				HIPFFT_Z2Z, useri.nthread_gpu);
			//hipfftExecZ2Z(tplan, dev_fft, dev_fft, HIPFFT_FORWARD);
#else
#error Unknown CUDA type selected
#endif
			hipDeviceSynchronize();

			alloc_status_im = hipMalloc(&dev_images_gpu, s_time_series.memory_one * capacity);
			alloc_status_corr_g = hipMalloc(&dev_corr_gpu, s_time_series.memory_one * useri.nthread_gpu);
			alloc_status_fftime = hipMalloc(&dev_fft_time_gpu, s_fft_time.memory_one * useri.nthread_gpu);
			alloc_status_pw = hipMalloc(&dev_power_spectra_gpu, s_power_spectra.memory_one );
			alloc_status_fft = hipMalloc(&dev_fft_gpu, s_fft.memory_tot);
			alloc_status_li = hipMalloc(&dev_im_gpu, s_load_image.memory_tot);
			alloc_status_rlut = hipMalloc(&dev_radial_lut_gpu, s_radial_lut.memory_tot);
			alloc_status_imsot = hipMalloc(&dev_image_sot_gpu, s_fft_images.memory_one);
			
			

		}



		//----------------------------------------------------------
		//----------------------------------------------------------
		//----------------------------------------------------------
		// this operation that seems unuseful in fact reset the plan in a such a way that, if ever some allocation errors
		// as occured, the program will work properly
		// this fact is purely experimental and I cannot explain why it happens!!!
				// CUFFT initialization
//#if (CUFFT_TYPE == CUFFT_TYPE_FLOAT)
//		hipfftExecC2C(plan, dev_fft, dev_fft, HIPFFT_FORWARD);
//#elif (CUFFT_TYPE == CUFFT_TYPE_DOUBLE)
//		hipfftExecZ2Z(plan, dev_fft_gpu, dev_fft_gpu, HIPFFT_FORWARD);
//#else
//#error Unknown CUDA type selected
//#endif
		//hipDeviceSynchronize();

		//----------------------------------------------------------
		//----------------------------------------------------------
		//----------------------------------------------------------
		hipError_t err = hipGetLastError();
		if (err != hipSuccess)
			printf("Error: %s\n", hipGetErrorString(err));

		//calc_sizes(1, dim_power_spectrum, capacity, sizeof(CUFFT_COMPLEX), s_fft_images);
		//calc_sizes(1, dim_power_spectrum, capacity, sizeof(STORE_REAL), s_power_spectra);

		// CUFFT initialization
#if (CUFFT_TYPE == CUFFT_TYPE_FLOAT)
		hipfftExecC2C(plan, dev_fft, dev_fft, HIPFFT_FORWARD);
#elif (CUFFT_TYPE == CUFFT_TYPE_DOUBLE)
		hipfftExecZ2Z(plan, dev_fft_gpu, dev_fft_gpu, HIPFFT_FORWARD);
#else
#error Unknown CUDA type selected
#endif
		hipDeviceSynchronize();

		err = hipGetLastError();
		if (err != hipSuccess)
			printf("Error: %s\n", hipGetErrorString(err));

		calc_sizes(1, nimages, capacity, sizeof(CUFFT_COMPLEX), s_time_series);

		tot_memory_fft_images = s_time_series.memory_tot;
		
		calc_sizes(1, dimtimeseries_zeropadding, useri.nthread_gpu, sizeof(CUFFT_COMPLEX), s_fft_time);
		tot_calculation_memory = s_fft_time.memory_tot;

		hipMemcpy(dev_radial_lut_gpu, ram_radial_lut, s_radial_lut.memory_one, hipMemcpyHostToDevice);

		err = hipGetLastError();
		if (err != hipSuccess)
			printf("Error: %s\n", hipGetErrorString(err));

		dev_images_cpu = new FFTW_COMPLEX[s_time_series.dim * capacity];
		dev_image_sot_cpu = new FFTW_COMPLEX[s_fft_images.dim];
		dev_power_spectra_cpu = new FFTW_REAL[s_power_spectra.dim];

		break;
	}
	default:
		break;
	}
	/*dev_radial_lut_gpu = new unsigned int[s_radial_lut.dim];
	for (i = 0; i < s_radial_lut.dim; ++i)
		dev_radial_lut_gpu[i] = ram_radial_lut[i];*/

	n_capacity = capacity;
	

	if(capacity == 0)	return 1;
	return 0;
}

void gpu_free_pointers()
{
	
	if(NULL != dev_im_gpu)
		{
			hipFree(dev_im_gpu);
			dev_im_gpu = NULL;
		}
	if(NULL != dev_fft_gpu)
		{
			hipFree(dev_fft_gpu);
			dev_fft_gpu = NULL;
		}
	if(NULL != dev_images_gpu)
		{
			hipFree(dev_images_gpu);
			dev_images_gpu = NULL;
		}
	if(NULL != dev_power_spectra_gpu )
		{
			hipFree(dev_power_spectra_gpu);
			dev_power_spectra_gpu = NULL;
		}
	if (NULL != dev_radial_lut_gpu)
	{
		hipFree(dev_radial_lut_gpu);
		dev_radial_lut_gpu = NULL;
	}

	if (NULL != dev_fft_time_gpu)
	{
		hipFree(dev_fft_time_gpu);
		dev_fft_time_gpu = NULL;
	}

	if (NULL != dev_corr_gpu)
	{
		hipFree(dev_corr_gpu);
		dev_corr_gpu = NULL;
	}

	if (NULL != dev_image_sot_gpu)
	{
		hipFree(dev_image_sot_gpu);
		dev_image_sot_gpu = NULL;
	}

	if (NULL != dev_image_sot_cpu)
	{
		delete[] dev_image_sot_cpu;
		dev_image_sot_cpu = NULL;
	}

	if (NULL != dev_images_cpu)
	{
		delete [] dev_images_cpu;
		dev_images_cpu = NULL;
	}

	if (NULL != dev_power_spectra_cpu)
	{
		delete[] dev_power_spectra_cpu;
		dev_power_spectra_cpu = NULL;
	}

}

void gpu_deallocation()
{
	gpu_free_pointers();
	hipfftDestroy(plan);
	hipfftDestroy(tplan);
}

void Image_to_complex_matrix(unsigned short* dev_im_gpu_, CUFFT_COMPLEX* dev_fft_gpu_, int i) {

	CUFFT_COMPLEX* dev_store_ptr;
	dev_store_ptr = &(dev_images_gpu[i * s_fft_images.dim]);

	time_fft_norm.start();
	short_to_real_with_gain << <s_load_image.cexe.nbk, s_load_image.cexe.nth >> >
		(s_load_image.dim, dev_im_gpu_, (CUFFT_REAL)(one_over_fft_norm), dev_fft_gpu_);

	hipfftExecZ2Z(plan, dev_fft_gpu_, dev_fft_gpu_, HIPFFT_FORWARD);

	hipDeviceSynchronize();


	CUFFT_REAL mean_tmp;
	STORE_REAL mean;
	// normalization
	hipMemcpy(&mean_tmp, dev_fft_gpu_, sizeof(CUFFT_REAL), hipMemcpyDeviceToHost);

	mean = mean_tmp;
	if (mean < 0.000000000000001)
	{
		mean = 1.;
		mean_tmp = 1.;
		//ret = 1;
		waitkeyboard(0);
	}
	mean_tmp = (CUFFT_REAL)(1. / mean_tmp);

	gain_complex_lut << <s_fft_images.cexe.nbk, s_fft_images.cexe.nth >> >
		(mean_tmp, s_fft_images.dim, dev_radial_lut_gpu, dev_fft_gpu_, dev_store_ptr);
	

	time_fft_norm.stop();

	hipDeviceSynchronize();
	/*CUFFT_COMPLEX* tmp_display_cpx_(NULL);

	tmp_display_cpx_ = new CUFFT_COMPLEX[s_fft_images.dim];

	hipMemcpy(tmp_display_cpx_, dev_store_ptr, s_fft_images.memory_one, hipMemcpyDeviceToHost);
	for (int ii = 0; ii < s_fft_images.dim; ++ii)
		std::cout << tmp_display_cpx_[ii].x << "  + i " << tmp_display_cpx_[ii].y << std::endl;*/

}

void Image_to_complex_matrix2(unsigned short* dev_im_gpu_, CUFFT_COMPLEX* dev_fft_gpu_, int i, INDEX nimages) {

	//CUFFT_COMPLEX* dev_store_ptr;
	//dev_store_ptr = &(dev_images_gpu[i * s_fft_images.dim]);
	//int alloc_status_im = hipMalloc(&dev_images_gpu1, nimages * s_power_spectra.dim * sizeof(CUFFT_COMPLEX));

	time_fft_norm.start();
	short_to_real_with_gain << <s_load_image.cexe.nbk, s_load_image.cexe.nth >> >
		(s_load_image.dim, dev_im_gpu_, (CUFFT_REAL)(one_over_fft_norm), dev_fft_gpu_);

	hipfftExecZ2Z(plan, dev_fft_gpu_, dev_fft_gpu_, HIPFFT_FORWARD);

	hipDeviceSynchronize();


	CUFFT_REAL mean_tmp;
	STORE_REAL mean;
	// normalization
	hipMemcpy(&mean_tmp, dev_fft_gpu_, sizeof(CUFFT_REAL), hipMemcpyDeviceToHost);

	mean = mean_tmp;
	if (mean < 0.000000000000001)
	{
		mean = 1.;
		mean_tmp = 1.;
		//ret = 1;
		waitkeyboard(0);
	}
	mean_tmp = (CUFFT_REAL)(1. / mean_tmp);

	gain_complex_lut_timeSeries << <s_fft_images.cexe.nbk, s_fft_images.cexe.nth >> >
		(i,nimages, mean_tmp, s_fft_images.dim, dev_radial_lut_gpu, dev_fft_gpu_, dev_images_gpu);


	time_fft_norm.stop();

	hipDeviceSynchronize();
	/*CUFFT_COMPLEX* tmp_display_cpx_(NULL);

	tmp_display_cpx_ = new CUFFT_COMPLEX[s_power_spectra.dim * nimages];

	hipMemcpy(tmp_display_cpx_, dev_images_gpu, nimages*s_fft_images.memory_one, hipMemcpyDeviceToHost);
	for (int ii = 0; ii < s_fft_images.dim* nimages; ++ii)
		std::cout << tmp_display_cpx_[ii].x << "  + i " << tmp_display_cpx_[ii].y << std::endl;

	FILE* version3;
	version3 = fopen("v33.txt", "w");
	for (int ii = 0; ii < nimages * s_power_spectra.dim; ++ii)
		//fprintf()
		fprintf(version3, "%d   %f    %f\n", ii, tmp_display_cpx_[ii].x, tmp_display_cpx_[ii].y);

	fclose(version3);*/

}
void Calc_StructureFunction_With_TimeCorrelation(INDEX nimages) {

	


	int alloc_status_corr_g = hipMalloc(&dev_corr_gpu1, nimages * s_power_spectra.dim*sizeof(CUFFT_REAL));

	int alloc_status_fftime = hipMalloc(&dev_fft_time_gpu1, s_fft_time.dim * s_power_spectra.dim * sizeof(CUFFT_COMPLEX));


	int threads = 1024;
	int blocksx = (s_power_spectra.dim + threads - 1) / threads;
	//int blocksx = (fft_size + threads - 1) / threads;
	//int blocksy1 = (nb_fft + threads1 - 2) / threads1;
	dim3 THREADS(threads);
	dim3 BLOCKS(blocksx);

	averagesabs2_array_gpu << < BLOCKS, THREADS >> > (nimages, s_power_spectra.dim, dev_images_gpu, dev_corr_gpu1);

	//CUFFT_COMPLEX* tmp_display_cpx_(NULL);
	/*CUFFT_REAL* dev_corr_cpu1(NULL);
	dev_corr_cpu1 = new CUFFT_REAL[s_power_spectra.dim * nimages];

	hipMemcpy(dev_corr_cpu1, dev_corr_gpu1, nimages * s_power_spectra.dim * sizeof(CUFFT_REAL), hipMemcpyDeviceToHost);
	FILE* version3;
	version3 = fopen("v22.txt", "w");
	for (int ii = 0; ii < nimages * s_power_spectra.dim; ++ii)
		//fprintf()
		fprintf(version3, "%d   %f \n", ii, dev_corr_cpu1[ii]);

	fclose(version3);*/
		//std::cout <<ii<<"   "<< dev_corr_cpu1[ii] << std::endl;
	int threads1 = 32;
	int blocksx1 = (nimages + threads1 - 1) / threads1;
	int blocksy1 = (s_power_spectra.dim  + threads1 - 1) / threads1;

	dim3 THREADS1(threads1, threads1);
	dim3 BLOCKS1(blocksx1, blocksy1);

	gaincomplex_gpu2 << <BLOCKS1, THREADS1 >> > (nimages, s_power_spectra.dim, s_fft_time.dim, dev_images_gpu,
		(FFTW_REAL)(1. / std::sqrt((FFTW_REAL)(s_fft_time.dim))), dev_fft_time_gpu1);

	hipDeviceSynchronize();

	
	hipfftExecZ2Z(tplan, dev_fft_time_gpu1, dev_fft_time_gpu1, HIPFFT_FORWARD);


	/*CUFFT_COMPLEX* dev_corr_cpu1(NULL);
	dev_corr_cpu1 = new CUFFT_COMPLEX[s_fft_time.dim * s_power_spectra.dim];

	hipMemcpy(dev_corr_cpu1, dev_fft_time_gpu1, s_fft_time.dim * s_power_spectra.dim * sizeof(CUFFT_COMPLEX), hipMemcpyDeviceToHost);
	FILE* version2;
	version2 = fopen("v222fft_.txt", "a");
	for (int ii = 0; ii < s_power_spectra.dim * s_fft_time.dim; ++ii)
		//fprintf()
		fprintf(version2, "%d   %f    %f \n", ii, dev_corr_cpu1[ii].x, dev_corr_cpu1[ii].y);

	fclose(version2);*/

	hipDeviceSynchronize();

	blocksx = (s_fft_time.dim * s_power_spectra.dim + threads - 1) / threads;
	dim3 THREADS2(threads);
	dim3 BLOCKS2(blocksx);
	complexabs2_gpu << <BLOCKS2, THREADS2 >> > (s_fft_time.dim * s_power_spectra.dim, dev_fft_time_gpu1, dev_fft_time_gpu1);

	hipDeviceSynchronize();

	hipfftExecZ2Z(tplan, dev_fft_time_gpu1, dev_fft_time_gpu1, HIPFFT_FORWARD);

	hipDeviceSynchronize();


	int threads2 = 32;
	int blocksx2 = (nimages + threads2 - 1) / threads2;
	int blocksy2 = (s_power_spectra.dim + threads2 - 1) / threads2;

	dim3 THREADS3(threads2, threads2);
	dim3 BLOCKS3(blocksx2, blocksy2);
	updatewithdivrebyramp_gpu2 << <BLOCKS3, THREADS3 >> > (nimages, s_power_spectra.dim, s_fft_time.dim, dev_fft_time_gpu1, dev_corr_gpu1, dev_power_spectra_gpu);

	CUFFT_REAL* dev_corr_cpu1(NULL);
	dev_corr_cpu1 = new CUFFT_REAL[s_power_spectra.dim * nimages];

	hipMemcpy(dev_corr_cpu1, dev_corr_gpu1, nimages * s_power_spectra.dim * sizeof(CUFFT_REAL), hipMemcpyDeviceToHost);
	FILE* version3;
	version3 = fopen("v222.txt", "w");
	for (int ii = 0; ii < nimages * s_power_spectra.dim; ++ii)
		//fprintf()
		fprintf(version3, "%d   %.10f \n", ii, dev_corr_cpu1[ii]);

	fclose(version3);

	std::cout << "test" << std::endl;


}

void Calc_structure_function(INDEX nimages,int i,int device_count) {


	int alloc_status_li_ = hipMalloc((void**)&dev_ALLfft_diff, s_power_spectra.memory_one * (nimages - 1));
	//cudaStatus = hipMalloc((void**)&dev_ALLfft_diff, lenght_ALLfft_diff * sizeof(int));
	if (alloc_status_li_ != hipSuccess) {
		fprintf(stderr, "hipMalloc dev_ALLfft_diff failed!");
		// goto Error;
	}
	lldiv_t group = std::div((long long)(nimages), (long long)(device_count));

	int n_group = (int)(group.quot);

	int  group_rem = (int)(group.rem);
	/*alloc_status_li_ = hipMalloc((void**)&dev_ALLpower_spectra, s_power_spectra.memory_one * nimages);
	if (alloc_status_li_ != hipSuccess) {
		fprintf(stderr, "hipMalloc dev_ALLpower_spectra failed!");
		// goto Error;
	}*/

	//Kernel 1 dim
	int threads = 32;
	int blocksx = (s_power_spectra.dim + threads - 1) / threads;
	int blocksy;// = (nb_fft + threads - 2) / threads;
   // dim3 THREADS(threads, threads);
   // dim3 BLOCKS(blocksx, blocksy);
	//int tot;

	//Kernel 2 dim
	int threads1 = 32;
	int blocksx1 = (s_power_spectra.dim + threads1 - 1) / threads1;
	//int blocksx = (fft_size + threads - 1) / threads;
	//int blocksy1 = (nb_fft + threads1 - 2) / threads1;
	dim3 THREADS1(threads1);
	dim3 BLOCKS1(blocksx1);
	hipError_t cudaStatus;

	//for (int z = i* (n_group+ group_rem); z < n_group + group_rem + i* n_group; z++) {
	for (int z = 0; z < nimages-1; z++) {


		//int threads = 32;
	    // int blocksx = (fft_size + threads - 1) / threads;
		blocksy = (nimages + threads - 2 - z) / threads;
		dim3 THREADS(threads, threads);
		dim3 BLOCKS(blocksx, blocksy);
		/*tot = 0;
		for (int len = 1; len <= z; len++) {
			tot += fft_size * (nb_fft - len);
		}*/
		time_differences.start();
		fft_diff << <BLOCKS, THREADS >> > (z, s_power_spectra.dim, nimages, dev_images_gpu, dev_ALLfft_diff);
		cudaStatus = hipDeviceSynchronize();
		
		structure_function << <BLOCKS1, THREADS1 >> > (z, s_power_spectra.dim, nimages, dev_ALLfft_diff, dev_power_spectra_gpu);
		
		time_differences.stop();
		/*cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			//goto Error;
		}*/
		//}
			// hipDeviceSynchronize waits for the kernel to finish, and returns
			// any errors encountered during the launch.
		//cudaStatus = hipDeviceSynchronize();

		/*if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			//goto Error;
		}*/

		// Copy output vector from GPU buffer to host memory.

		/*cudaStatus = hipMemcpy(ALLfft_diff_GPU, dev_ALLfft_diff, fft_size * (nb_fft - 1) * sizeof(int), hipMemcpyDeviceToHost);
		//cudaStatus = hipMemcpy(ALLfft_diff_GPU, dev_ALLfft_diff, lenght_ALLfft_diff * sizeof(int), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			//goto Error;
		}

		std::cout << std::endl << std::endl;*/
		// int tot = 0;



		/*for (int ii = 0; ii < nb_fft - 1 - z; ii++) {
			for (int j = 0; j < fft_size; j++) {
				//ALLfft_diff[j + ii * fft_size] = ALLfft[j + ii * fft_size] - ALLfft[j + fft_size + ii * fft_size];
				std::cout << ALLfft_diff_GPU[j + ii * fft_size] << "   ";
			}
			std::cout << std::endl;
		}*/

		//std::cout << std::endl << std::endl;

		//std::cout << std::endl << std::endl;

		// exit(0);
		 //int z = 0;
		

		


		/*cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addKerne2 launch failed: %s\n", hipGetErrorString(cudaStatus));
			//goto Error;
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			//goto Error;
		}*/



		// Copy output vector from GPU buffer to host memory.
		/*cudaStatus = hipMemcpy(power_spectra_GPU, dev_ALLpower_spectra, s_power_spectra.memory_one * (nimages - 1), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			//goto Error;
		}
		//std::cout << std::endl << std::endl;
		//for (int z = 0; z < nb_fft - 1; z++) {

		std::cout << "d(" << z + 1 << ") =" << std::endl;
		for (int k = 0; k < fft_size; k++) {

			std::cout << power_spectra_GPU[k + z * fft_size] << "   ";
		}

		std::cout << std::endl << "-------------------------------------" << std::endl;*/
	}


}

int image_to_dev_gpu(SIGNED_INDEX ind_fifo, STORE_REAL &mean, unsigned short *im, bool flg_debug)
{
	CUFFT_COMPLEX* dev_store_ptr;

	// selecting the correct memory area
	if (0 > ind_fifo)
		dev_store_ptr = dev_image_sot_gpu;
	else
		dev_store_ptr = &(dev_images_gpu[ind_fifo * s_fft_images.dim]);

	if (true == flg_debug)
	{
		double *img[2];
		if (NULL == image_l) image_l = new double[s_load_image.dim];
		if ((NULL == module_fft) && (NULL == phase_fft) && (NULL == tmp_display_i16) && (NULL == tmp_display_cpx))
		{
			module_fft = new double[s_load_image.dim];
			phase_fft = new double[s_load_image.dim];
			img[0] = module_fft; img[1] = phase_fft;
			tmp_display_i16 = new unsigned __int16[s_load_image.dim];
			tmp_display_cpx = new CUFFT_COMPLEX[s_load_image.dim];
		}
		if ((NULL == fft_window) && (NULL == image_window))
		{
			bool flg_colormap = true;
			fft_window = new_figure(s_load_image.dimx, s_load_image.dimy, 2, img);
			image_window = new_figure(s_load_image.dimx, s_load_image.dimy, image_l);
			fft_window->control[ID_BUTTON_COLORMAP]->set_value(&flg_colormap);
			fft_window->colormap = cv::COLORMAP_HSV;
		}
	}

	INDEX i;
	int ret = 0;
	CUFFT_REAL mean_tmp;

	// memory copy from RAM to video card
	time_from_host_to_device.start();
	hipMemcpy( dev_im_gpu, im, s_load_image.memory_one, hipMemcpyHostToDevice);
	time_from_host_to_device.stop();

	

	if (true == flg_debug)
	{
		hipMemcpy(tmp_display_i16, dev_im_gpu, s_load_image.memory_one, hipMemcpyDeviceToHost);
		for (i = 0; i < s_load_image.dim; ++i) image_l[i] = (double)(tmp_display_i16[i]);
		image_window->show();
		waitkeyboard();
	}

 time_fft_norm.start();



	// from image to complex matrix
	short_to_real_with_gain<<<s_load_image.cexe.nbk, s_load_image.cexe.nth>>>
		                      (s_load_image.dim ,dev_im_gpu, (CUFFT_REAL)(one_over_fft_norm), dev_fft_gpu);

	hipDeviceSynchronize();
	if (true == flg_debug)
	{
		hipMemcpy(tmp_display_cpx, dev_fft_gpu, s_fft.memory_one, hipMemcpyDeviceToHost);
		for (i = 0; i < s_load_image.dim; ++i)	image_l[i] = tmp_display_cpx[i].x;
		complex_to_module_phase(s_load_image.dim, tmp_display_cpx, module_fft, phase_fft);
		image_window->show();
		fft_window->show();
		waitkeyboard();
	}



	// FFT execution
	#if (CUFFT_TYPE == CUFFT_TYPE_FLOAT)
	 hipfftExecC2C(plan, dev_fft, dev_fft, HIPFFT_FORWARD);
//		hipfftExecC2C(plan, dev_fft, dev_fft, HIPFFT_FORWARD);
	#elif (CUFFT_TYPE == CUFFT_TYPE_DOUBLE)
	 hipfftExecZ2Z(plan, dev_fft_gpu, dev_fft_gpu, HIPFFT_FORWARD);
	#else
		#error Unknown CUDA type selected
	#endif


	hipDeviceSynchronize();



	// normalization
	hipMemcpy( &mean_tmp, dev_fft_gpu, sizeof(CUFFT_REAL), hipMemcpyDeviceToHost);

	mean = mean_tmp;
	if(mean < 0.000000000000001)
		{
			mean = 1.;
			mean_tmp = 1.;
			ret = 1;
			waitkeyboard(0);
		}
	mean_tmp =	(CUFFT_REAL)(1./mean_tmp);
	
	//mean_tmp = 1.0;
	//gain_complex<<<s_fft_images.cexe.nbk, s_fft_images.cexe.nth>>>(mean_tmp,s_fft_images.dim, dev_fft, dev_store_ptr);
	gain_complex_lut<<<s_fft_images.cexe.nbk, s_fft_images.cexe.nth >>>
		(mean_tmp, s_fft_images.dim, dev_radial_lut_gpu, dev_fft_gpu, dev_store_ptr);
	hipDeviceSynchronize();
	time_fft_norm.stop();

	

	if (true == flg_debug)
	{
		hipMemcpy(tmp_display_cpx, dev_fft_gpu, s_fft.memory_one, hipMemcpyDeviceToHost);
		complex_to_module_phase(s_load_image.dim, tmp_display_cpx, module_fft, phase_fft);
		module_fft[0] = 0.0;
		image_window->show();
		fft_window->show();
		waitkeyboard(5);

		hipMemcpy(tmp_display_cpx, dev_store_ptr, s_fft_images.memory_one, hipMemcpyDeviceToHost);
		memset(module_fft, 0, s_fft.dim * sizeof(double));
		memset(phase_fft, 0, s_fft.dim * sizeof(double));
		for (i = 0; i < s_fft_images.dim; ++i)
		{
			module_fft[ram_radial_lut[i]] = sqrt(tmp_display_cpx[i].x*tmp_display_cpx[i].x + tmp_display_cpx[i].y*tmp_display_cpx[i].y);
			phase_fft[ram_radial_lut[i]] = atan2(tmp_display_cpx[i].y, tmp_display_cpx[i].x);
			module_fft[ram_radial_lut[i]] = tmp_display_cpx[i].x;
		}
		/*
		for (i = 0; i < s_fft_images.dim; ++i)
		{
			module_fft[ram_radial_lut[i]] = i+0.1;
			phase_fft[ram_radial_lut[i]] = 0;
		}
		hipMemcpy(ram_radial_lut, dev_radial_lut, s_fft_images.dim*sizeof(unsigned int), hipMemcpyDeviceToHost);
		for (i = 0; i < s_fft_images.dim; ++i)
		{
			module_fft[ram_radial_lut[i]] -= i;
	//		phase_fft[ram_radial_lut[i]] = i;
		}*/
		fft_window->show();
		waitkeyboard(5);
	}

	++n_computed_fft;

	

	return ret;
}

void copy_power_spectra_from_dev_gpu(STORE_REAL *power_spectrum_r)
{
	int version3 = 0;
	if (version3 == 0) {
		hipMemcpy(power_spectrum_r, dev_power_spectra_gpu, s_power_spectra.memory_tot, hipMemcpyDeviceToHost);
	}
	else {

		hipMemcpy(power_spectrum_r, dev_power_spectra_gpu, s_power_spectra.memory_tot, hipMemcpyDeviceToHost);

	}
}


void diff_power_spectrum_to_avg_gpu_gpu(CUFFT_REAL coef1, CUFFT_REAL coef2, INDEX j, INDEX ind_dist)
{
	//std::cout << ind_dist <<"   "<< coef1<<"   "<< coef2<< std::endl;

	diff_power_spectrum_to_avg_gpu << <s_power_spectra.cexe.nbk, s_power_spectra.cexe.nth >> >
		(s_power_spectra.dim, &(dev_images_gpu[j * s_fft_images.dim]), dev_image_sot_gpu, coef1, coef2, &(dev_power_spectra_gpu[ind_dist * s_power_spectra.dim]));

	//sync
	hipDeviceSynchronize();
}




void timeseriesanalysis_gpu(INDEX dimtimeseries, INDEX dim_t, CUFFT_COMPLEX* tseries, INDEX dimfft, CUFFT_COMPLEX* fft_memory, hipfftHandle* tplan, CUFFT_REAL* corr_memory, cuda_exec mycuda_dim_t, cuda_exec mycuda_dim, cuda_exec mycuda_dim_dim_t)
{
	INDEX i;

	
	cuda_exec mycuda_dim_p;
	cuda_exec mycuda_dim_dim_p;

	calc_cuda_exec(dimfft, deviceProp.maxThreadsPerBlock, &mycuda_dim_p);
	calc_cuda_exec(dimfft* dim_t, deviceProp.maxThreadsPerBlock, &mycuda_dim_dim_p);

	

	//----------------------------------------------
	// Calculating the average of the absolute squares
	averagesabs2_array_gpu << < mycuda_dim_t.nbk, mycuda_dim_t.nth >> > (dimtimeseries,dim_t, tseries, corr_memory);

	/*CUFFT_REAL* dev_corr_cpu1(NULL);
	dev_corr_cpu1 = new CUFFT_REAL[s_time_series.dim * useri.nthread_gpu];

	hipMemcpy(dev_corr_cpu1, corr_memory, s_time_series.dim * useri.nthread_gpu *sizeof(CUFFT_REAL), hipMemcpyDeviceToHost);
	FILE* version2;
	version2 = fopen("v2.txt", "a");
	for (int ii = 0; ii < useri.nthread_gpu * s_time_series.dim; ++ii)
		//fprintf()
		fprintf(version2, "%d   %f \n", ii , dev_corr_cpu1[ii]);

	fclose(version2);*/
	

	//----------------------------------------------
	// preparing Fourier variables

	//-----------------------------------------------
	// zeroing all fft memory for zero padding and copying data with normalization to fft memory
	/*for (i = 0; i < dimfft; ++i)
	{
		fft_memory[i].x = 0.0;
		fft_memory[i].y = 0.0;
	}*/

	gaincomplex_gpu << <mycuda_dim_dim_p.nbk, mycuda_dim_dim_p.nth >> > (dimfft * dim_t, fft_memory, 0.0, fft_memory);

	hipDeviceSynchronize();

	//std::cout << hipGetLastError() << std::endl;
	for (i = 0; i < dim_t; ++i)
	{
		gaincomplex_gpu << <mycuda_dim.nbk, mycuda_dim.nth >> > (dimtimeseries, &tseries[i * dimtimeseries], (FFTW_REAL)(1. / std::sqrt((FFTW_REAL)(dimfft))), &fft_memory[i * dimfft]);
		//----------------------------------------------

		hipDeviceSynchronize();
		//std::cout << hipGetLastError() << std::endl;
	}

	/*CUFFT_COMPLEX* dev_corr_cpu1(NULL);
	dev_corr_cpu1 = new CUFFT_COMPLEX[dimfft * useri.nthread_gpu];

	hipMemcpy(dev_corr_cpu1, fft_memory, dimfft * useri.nthread_gpu * sizeof(CUFFT_COMPLEX), hipMemcpyDeviceToHost);
	FILE* version2;
	version2 = fopen("v2.txt", "a");
	for (int ii = 0; ii < useri.nthread_gpu * dimfft; ++ii)
		//fprintf()
		fprintf(version2, "%d   %f    %f \n", ii, dev_corr_cpu1[ii].x, dev_corr_cpu1[ii].y);

	fclose(version2);*/
		// FFT execution
		// FFT execution
		#if (CUFFT_TYPE == CUFFT_TYPE_FLOAT)
				hipfftExecC2C(tplan[0], fft_memory, fft_memory, HIPFFT_FORWARD);
				//		hipfftExecC2C(plan, dev_fft, dev_fft, HIPFFT_FORWARD);
		#elif (CUFFT_TYPE == CUFFT_TYPE_DOUBLE)
				hipfftExecZ2Z(tplan[0], fft_memory, fft_memory, HIPFFT_FORWARD);
		#else
		#error Unknown CUDA type selected
		#endif

		hipDeviceSynchronize();
		//std::cout << hipGetLastError() << std::endl;

		/*CUFFT_COMPLEX* dev_corr_cpu1(NULL);
		dev_corr_cpu1 = new CUFFT_COMPLEX[dimfft * useri.nthread_gpu];

		hipMemcpy(dev_corr_cpu1, fft_memory, dimfft * useri.nthread_gpu * sizeof(CUFFT_COMPLEX), hipMemcpyDeviceToHost);
		FILE* version2;
		version2 = fopen("v21.txt", "a");
		for (int ii = 0; ii < dimfft * useri.nthread_gpu; ++ii)
			//fprintf()
			fprintf(version2, "%d   %f    %f \n", ii, dev_corr_cpu1[ii].x, dev_corr_cpu1[ii].y);

		fclose(version2);*/


		//fftw_execute(tplan[0]);

		// evaluating abs^2
		// change my cuda to bigger one
		complexabs2_gpu << <mycuda_dim_dim_p.nbk, mycuda_dim_dim_p.nth >> > (dimfft * dim_t, fft_memory, fft_memory);
		hipDeviceSynchronize();
		//std::cout << hipGetLastError() << std::endl;
		// FFT execution. Given the conditions
		// - we are only interested in the real part
		// - we start from a real function generated by the absolute value
		// Then the direct and inverse FFT are equivalent. We can re-use plan_direct!

		#if (CUFFT_TYPE == CUFFT_TYPE_FLOAT)
				hipfftExecC2C(tplan[0], fft_memory, fft_memory, HIPFFT_FORWARD);
				//		hipfftExecC2C(plan, dev_fft, dev_fft, HIPFFT_FORWARD);
		#elif (CUFFT_TYPE == CUFFT_TYPE_DOUBLE)
				hipfftExecZ2Z(tplan[0], fft_memory, fft_memory, HIPFFT_FORWARD);
		#else
		#error Unknown CUDA type selected
		#endif

		hipDeviceSynchronize();
		//std::cout << hipGetLastError() << std::endl;
for (i = 0; i < dim_t; ++i)
{
		updatewithdivrebyramp_gpu << <mycuda_dim.nbk, mycuda_dim.nth >> > (dimtimeseries, dimtimeseries, &fft_memory[i * dimfft], &corr_memory[i * dimtimeseries]);
		// copy the result  back to original memory area

		hipDeviceSynchronize();
		//std::cout << hipGetLastError() << std::endl;

		copyfrom_gpu << <mycuda_dim.nbk, mycuda_dim.nth >> > (dimtimeseries, &tseries[i * dimtimeseries], &corr_memory[i * dimtimeseries]);
		hipDeviceSynchronize();
		//std::cout << hipGetLastError() << std::endl;
		/*for (INDEX	j = 0; j < dimtimeseries; ++j)
		{
			tseries[j+ i * dimtimeseries].x = corr_memory[j + i * dimtimeseries];
			tseries[j + i * dimtimeseries].y = 0.0;
		}*/
	}

CUFFT_REAL* dev_corr_cpu1(NULL);
dev_corr_cpu1 = new CUFFT_REAL[s_time_series.dim * useri.nthread_gpu];

hipMemcpy(dev_corr_cpu1, corr_memory, s_time_series.dim * useri.nthread_gpu * sizeof(CUFFT_REAL), hipMemcpyDeviceToHost);
FILE* version2;
version2 = fopen("v22.txt", "a");
for (int ii = 0; ii < useri.nthread_gpu * s_time_series.dim; ++ii)
	//fprintf()
	fprintf(version2, "%d   %.10f \n", ii, dev_corr_cpu1[ii]);

fclose(version2);

}


